#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "dll.h"

#define G  6.67408e-11

using namespace std;
namespace nbodydll
{
	__global__ void cuda_integrate(double* x, double* y, double* z, double* xacc, double* yacc, double* zacc, double* vx, double* vy, double*vz,
		double* md, double* m, double*datastorage, int cur_iteration, double timedelta, int cp)
	{
		int cur_body = blockIdx.x*blockDim.x + threadIdx.x;

		if (cur_body < cp)
		{
			xacc[cur_body] = 0;
			yacc[cur_body] = 0;
			zacc[cur_body] = 0;

			for (int i = 0; i < cp; i++)
			{
				if (i != cur_body)
				{
					xacc[cur_body] += (x[i] - x[cur_body]) * G * m[i] / ((md[i*cp + cur_body] * md[i*cp + cur_body]) * md[i*cp + cur_body]);
					yacc[cur_body] += (y[i] - y[cur_body]) * G* m[i] / ((md[i*cp + cur_body] * md[i*cp + cur_body]) * md[i*cp + cur_body]);
					zacc[cur_body] += (z[i] - z[cur_body]) * G * m[i] / ((md[i*cp + cur_body] * md[i*cp + cur_body]) * md[i*cp + cur_body]);
				}
			}

			x[cur_body] = x[cur_body] + vx[cur_body] * timedelta + xacc[cur_body] * timedelta*timedelta / 2;
			y[cur_body] = y[cur_body] + vy[cur_body] * timedelta + yacc[cur_body] * timedelta*timedelta / 2;
			z[cur_body] = z[cur_body] + vz[cur_body] * timedelta + zacc[cur_body] * timedelta*timedelta / 2;
			vx[cur_body] += xacc[cur_body] * timedelta;
			vy[cur_body] += yacc[cur_body] * timedelta;
			vz[cur_body] += zacc[cur_body] * timedelta;

			datastorage[(cur_iteration*cp + cur_body) * 3] = x[cur_body];
			datastorage[(cur_iteration*cp + cur_body) * 3 + 1] = y[cur_body];
			datastorage[(cur_iteration*cp + cur_body) * 3 + 2] = z[cur_body];
		}
	}

	__global__ void cuda_matrix_calculate(double* md, double* x, double* y, double* z, int cp)
	{
		int cur_body = blockIdx.x * blockDim.x + threadIdx.x;
		if (cur_body < cp)
		{
			for (int i = 0; i < cp; i++)
			{
				md[cur_body*cp + i] = md[i*cp + cur_body] = sqrt((x[cur_body] - x[i]) * (x[cur_body] - x[i]) + (y[cur_body] - y[i]) * (y[cur_body] - y[i])
					+ (z[cur_body] - z[i]) * (z[cur_body] - z[i]));
			}
		}

	}

	void cudaIntegrate(int* cp, double* x, double* y, double* z, double* vx, double* vy, double*vz, double* m, int iterations,
		double*datastorage, double* time, double startenergy, bool* merge, double* mergenergy)
	{

		//host
		double * md = new double[(*cp)*(*cp)];

		//device
		double * xdev;
		double *ydev;
		double *zdev;
		double *vxdev;
		double* vydev;
		double* vzdev;
		double* xaccdev;
		double* yaccdev;
		double* zaccdev;
		double* mddev;
		double* datastoragedev;
		double* mdev;

		hipMalloc((double**)&xdev, (*cp) * sizeof(double));
		hipMalloc((double**)&ydev, (*cp) * sizeof(double));
		hipMalloc((double**)&zdev, (*cp) * sizeof(double));
		hipMalloc((double**)&vxdev, (*cp) * sizeof(double));
		hipMalloc((double**)&vydev, (*cp) * sizeof(double));
		hipMalloc((double**)&vzdev, (*cp) * sizeof(double));
		hipMalloc((double**)&xaccdev, (*cp) * sizeof(double));
		hipMalloc((double**)&yaccdev, (*cp) * sizeof(double));
		hipMalloc((double**)&zaccdev, (*cp) * sizeof(double));
		hipMalloc((double**)&mdev, (*cp) * sizeof(double));
		hipMalloc((double**)&mddev, (*cp) * (*cp) * sizeof(double));
		hipMalloc((double**)&datastoragedev, (*cp) * (iterations + 1) * 3 * sizeof(double));


		hipMemcpy(xdev, x, (*cp) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(ydev, y, (*cp) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(zdev, z, (*cp) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vxdev, vx, (*cp) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vydev, vy, (*cp) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vzdev, vz, (*cp) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(mdev, m, (*cp) * sizeof(double), hipMemcpyHostToDevice);

		double deltatime = 0;
		int numberofblocks = (*cp) / 128 + 1;
		time[0] = 0;
		cuda_matrix_calculate << <numberofblocks, 128 >> > (mddev, xdev, ydev, zdev, (*cp));
		for (int i = 0; i <= iterations; i++)
		{
			if (i % 25 == 0)
			{
				hipMemcpy(md, mddev, (*cp) * (*cp) * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(vx, vxdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(vy, vydev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(vz, vzdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);

				double kinetic_energy = 0;
				for (int i = 0; i < (*cp); i++)
				{
					kinetic_energy += m[i] * (vx[i] * vx[i] + vy[i] * vy[i] + vz[i] * vz[i]) / 2;
				}
				double potential_energy = 0;
				for (int i = 0; i < (*cp); i++)
				{
					for (int j = i + 1; j < (*cp); j++)
					{
						potential_energy -= G*m[i] * m[j] / md[i*(*cp) + j];
					}
				}
				double new_total_energy = kinetic_energy + potential_energy + *mergenergy;
				double delta_energy = startenergy - new_total_energy;
				double coeff = sqrt(1 + delta_energy / kinetic_energy);
				for (int i = 0; i < (*cp); i++)
				{
					vx[i] *= coeff;
					vy[i] *= coeff;
					vz[i] *= coeff;
				}

				double x_impulse = 0;
				double y_impulse = 0;
				double z_impulse = 0;
				double sum_mass = 0;

				for (int i = 0; i < (*cp); i++)
				{
					x_impulse += m[i] * vx[i];
					y_impulse += m[i] * vy[i];
					z_impulse += m[i] * vz[i];
					sum_mass += m[i];
				}

				x_impulse = x_impulse / sum_mass;
				y_impulse = y_impulse / sum_mass;
				z_impulse = z_impulse / sum_mass;

				for (int i = 0; i < (*cp); i++)
				{
					vx[i] -= x_impulse;
					vy[i] -= y_impulse;
					vz[i] -= z_impulse;
				}

				double mindist = 1e290;
				int indexi = 0;
				int indexj = 0;
				for (int i = 0; i < (*cp); i++)
				{
					for (int j = i + 1; j < (*cp); j++)
					{
						if (md[i*(*cp) + j] < mindist)
						{
							indexi = i;
							indexj = j;
							mindist = md[i*(*cp) + j];
						}
					}
				}
				double maxv = 0;
				if (mindist < 1e9)
				{

					hipMemcpy(md, mddev, (*cp)*(*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(x, xdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(y, ydev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(z, zdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(vx, vxdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(vy, vydev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(vz, vzdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
					hipMemcpy(datastorage, datastoragedev, (*cp)*(iterations + 1) * 3 * sizeof(double), hipMemcpyDeviceToHost);

					*merge = true;
					*mergenergy = *mergenergy - G*m[indexi] * m[indexj] / mindist;
					vx[indexi] = (m[indexi] * vx[indexi] + m[indexj] * vx[indexj]) / (m[indexi] + m[indexj]);
					vy[indexi] = (m[indexi] * vy[indexi] + m[indexj] * vy[indexj]) / (m[indexi] + m[indexj]);
					vz[indexi] = (m[indexi] * vz[indexi] + m[indexj] * vz[indexj]) / (m[indexi] + m[indexj]);

					x[indexi] = (m[indexi] * x[indexi] + m[indexj] * x[indexj]) / (m[indexi] + m[indexj]);
					y[indexi] = (m[indexi] * y[indexi] + m[indexj] * y[indexj]) / (m[indexi] + m[indexj]);
					z[indexi] = (m[indexi] * z[indexi] + m[indexj] * z[indexj]) / (m[indexi] + m[indexj]);

					m[indexi] = m[indexi] + m[indexj];

					for (int i = indexj; i < (*cp) - 1; i++)
					{
						x[i] = x[i + 1];
						y[i] = y[i + 1];
						z[i] = z[i + 1];
						vx[i] = vx[i + 1];
						vy[i] = vy[i + 1];
						vz[i] = vz[i + 1];
					}
					(*cp) = (*cp) - 1;

					for (int k = i; k < iterations + 1; k++)
					{
						time[k + 1] = time[i];
					}

					for (int k = i; k < iterations + 1; k++)
					{
						for (int j = 0; j < (*cp) + 1; j++)
						{
							datastorage[3 * (k*(*cp) + j)] = x[j];
							datastorage[3 * (k*(*cp) + j) + 1] = y[j];
							datastorage[3 * (k*(*cp) + j) + 2] = z[j];
						}

					}
					break;
				}

				maxv = sqrt(max(vx[indexi] * vx[indexi] + vy[indexi] * vy[indexi] + vz[indexi] * vz[indexi], vx[indexj] * vx[indexj] + vy[indexj] * vy[indexj] + vz[indexj] * vz[indexj]));
				deltatime = 1e-2*mindist / maxv;

				hipMemcpy(vxdev, vx, (*cp) * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(vydev, vy, (*cp) * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(vzdev, vz, (*cp) * sizeof(double), hipMemcpyHostToDevice);
			}
			if (i != 0)
			{
				time[i] = time[i - 1] + deltatime;
			}
			cuda_integrate << <numberofblocks, 128 >> > (xdev, ydev, zdev, xaccdev, yaccdev, zaccdev, vxdev, vydev, vzdev, mddev, mdev, datastoragedev, i, deltatime, (*cp));
			cuda_matrix_calculate << <numberofblocks, 128 >> > (mddev, xdev, ydev, zdev, (*cp));
		}
		if (!*merge)
		{
			hipMemcpy(md, mddev, (*cp)*(*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(x, xdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(y, ydev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(z, zdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(vx, vxdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(vy, vydev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(vz, vzdev, (*cp) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(datastorage, datastoragedev, (*cp)*(iterations + 1) * 3 * sizeof(double), hipMemcpyDeviceToHost);
		}


		delete[] md;

		hipFree(xdev);
		hipFree(ydev);
		hipFree(zdev);
		hipFree(vxdev);
		hipFree(vydev);
		hipFree(vzdev);
		hipFree(xaccdev);
		hipFree(yaccdev);
		hipFree(zaccdev);
		hipFree(mdev);
		hipFree(mddev);
		hipFree(datastoragedev);


	}
}
